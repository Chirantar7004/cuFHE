/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

int main() {
  hipSetDevice(1);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  uint32_t kNumTests = kNumSMs * 32;

  SetSeed(); // set random seed

  PriKey pri_key_old; // private key
  PubKey pub_key_old; // public key
  Ptxt* pt = new Ptxt[2 * kNumTests];
  Ctxt* ct = new Ctxt[2 * kNumTests];
  Synchronize();
  bool correct;

  cout<< "------ Key Generation ------" <<endl;
  KeyGen(pub_key_old, pri_key_old);
  // Alternatively ...
  // PriKeyGen(pri_key);
  // PubKeyGen(pub_key, pri_key);
  WritePriKeyToFile(pri_key_old, "pri_key.txt");
  WritePubKeyToFile(pub_key_old, "pub_key.txt");
  PriKey pri_key; // private key
  PubKey pub_key; // public key
  ReadPriKeyFromFile(pri_key, "pri_key.txt");
  ReadPubKeyFromFile(pub_key, "pub_key.txt");

  cout<< "------ Test Encryption/Decryption ------" <<endl;
  cout<< "Number of tests:\t" << kNumTests <<endl;
  correct = true;
  for (int i = 0; i < kNumTests; i ++) {
    pt[i].message_ = rand() % Ptxt::kPtxtSpace;
    Encrypt(ct[i], pt[i], pri_key);
    Decrypt(pt[kNumTests + i], ct[i], pri_key);
    if (pt[kNumTests + i].message_ != pt[i].message_) {
      correct = false;
      break;
    }
  }
  if (correct)
    cout<< "PASS" <<endl;
  else
    cout<< "FAIL" <<endl;

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing

  cout<< "------ Test NAND Gate ------" <<endl;
  cout<< "Number of tests:\t" << kNumTests <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();

  correct = true;
  for (int i = 0; i < 2 * kNumTests; i ++) {
    pt[i] = rand() % Ptxt::kPtxtSpace;
    Encrypt(ct[i], pt[i], pri_key);
  }
  Synchronize();

  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Here, pass streams to gates for parallel gates.
  for (int i = 0; i < kNumTests; i ++)
    Nand(ct[i], ct[i], ct[i + kNumTests], st[i % kNumSMs]);
  Synchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);
  cout<< et / kNumTests << " ms / gate" <<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);
  int num_failures = 0;
  for (int i = 0; i < kNumTests; i ++) {
    NandCheck(pt[i + kNumTests], pt[i], pt[i + kNumTests]);
/*    hipMemcpy(ct[i].lwe_sample_->data(),
               ct[i].lwe_sample_device_->data(),
               ct[i].lwe_sample_->SizeData(),
               hipMemcpyDeviceToHost);*/
    Decrypt(pt[i], ct[i], pri_key);
    if (pt[i + kNumTests].message_ != pt[i].message_) {
      correct = false;
      num_failures ++;//break;
    }
  }
  if (correct)
    cout<< "PASS" <<endl;
  else
    cout<< "FAIL\t" << num_failures <<endl;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  delete [] st;
  return 0;
}
