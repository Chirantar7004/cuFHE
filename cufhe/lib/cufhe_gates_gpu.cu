/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <include/cufhe_gpu.cuh>
#include <include/bootstrap_gpu.cuh>
#include <include/details/error_gpu.cuh>

namespace cufhe {

void Initialize(const PubKey& pub_key) {
  BootstrappingKeyToNTT(pub_key.bk_);
  KeySwitchingKeyToDevice(pub_key.ksk_);
}

void CleanUp() {
  DeleteBootstrappingKeyNTT();
  DeleteKeySwitchingKey();
}

inline void CopyCtxtH2D(const Ctxt& c, Stream st) {
  CuSafeCall(hipMemcpyAsync(c.lwe_sample_device_->data(),
                             c.lwe_sample_->data(),
                             c.lwe_sample_->SizeMalloc(),
                             hipMemcpyHostToDevice,
                             st.st()));
}

inline void CopyCtxtD2H(const Ctxt& c, Stream st) {
  CuSafeCall(hipMemcpyAsync(c.lwe_sample_->data(),
                             c.lwe_sample_device_->data(),
                             c.lwe_sample_->SizeMalloc(),
                             hipMemcpyDeviceToHost,
                             st.st()));
}

//void Initialize(PubKey pub_key);
//void And (Ctxt& out, const Ctxt& in0, const Ctxt& in1, const PubKey& pub_key);
//void Or  (Ctxt& out, const Ctxt& in0, const Ctxt& in1, const PubKey& pub_key);
//void Xor (Ctxt& out, const Ctxt& in0, const Ctxt& in1, const PubKey& pub_key);
void Nand(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - in0.lwe_sample_->data()[i]
                                   - in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;

  CopyCtxtH2D(out, st);
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
  CopyCtxtD2H(out, st);
}

void Or(Ctxt& out,
        const Ctxt& in0,
        const Ctxt& in1,
        Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + in0.lwe_sample_->data()[i]
                                   + in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void And(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + in0.lwe_sample_->data()[i]
                                   + in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Nor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - in0.lwe_sample_->data()[i]
                                   - in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Xor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 4);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + 2 * in0.lwe_sample_->data()[i]
                                   + 2 * in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Xnor(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 4);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - 2 * in0.lwe_sample_->data()[i]
                                   - 2 * in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Not(Ctxt& out,
         const Ctxt& in,
         Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = -in.lwe_sample_->data()[i];
}

void Copy(Ctxt& out,
          const Ctxt& in,
          Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = in.lwe_sample_->data()[i];
}

} // namespace cufhe
